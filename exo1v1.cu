#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BSIZE 1024

__global__
void kernel(uint* dvec, uint size, uint* dres)
{


    uint x = threadIdx.x;
    if (x< size)
    {
        for (uint i = 1; i < size; i*=2)
        {
            if (x%2*i == 0)
            {
              printf("THREAD N° %u : Somme avant %u\n", x, *dres);
              printf("THREAD N° %u : Elem du vecteur : %u\n", x, dvec[x]);
              dvec[x]+=dvec [x+i];
            }
            __syncthreads(); //waiting for all the other threads to finish
        }       
    }
        printf("THREAD N° %u : Somme actuelle %u\n", x, *dres);
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: <filename>\n");
        exit(-1);
    }
    unsigned int log2size, size;
    unsigned int *vec;
    FILE *f = fopen(argv[1], "r");
    fscanf(f, "%d\n", &log2size);
    if (log2size > 10) {
        printf("Size (%u) is too large: size is limited to 2^10\n", log2size);
        exit(-1);
    }
    size = 1 << log2size; // taille du tab
    unsigned int bytes = size * sizeof(unsigned int); // taille des élém du tableau
    vec = (unsigned int *) malloc(bytes); // notre vecteur d'éléments
    assert(vec);
    for (unsigned int i = 0; i < size; i++) {
        fscanf(f, "%u\n", &(vec[i])); // ajout des élem dans vec
    }
    fclose(f);

    uint* dres = 0; // res de la somme on device
    uint res;
    uint* dvec;

    hipError_t err = hipMalloc((void**)&dvec,bytes); //reserve la taille du tab dans la mémoire 
    if (err != hipSuccess){
        printf("%s in %s at line %d\n",
        hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(dvec, vec, bytes, hipMemcpyHostToDevice);
    // dst, src, byte, kind of copy

    err = hipMalloc((void**)&dres, sizeof(int)); // alloc taille res
    if (err != hipSuccess){
        printf("%s in %s at line %d\n",
        hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    kernel<<<(size +1023 )/BSIZE, BSIZE>>>(dvec, size, dres);

    hipMemcpy(&res, dres, sizeof(int), hipMemcpyDeviceToHost);
    // dst, src, byte, kind of copy

    // Objectif : obtenir 34 avec file1
    printf("Somme = %u", res);

    hipFree(dvec);
    hipFree(dres);
}
