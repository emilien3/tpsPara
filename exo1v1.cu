#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BSIZE 1024

__global__
void kernel(uint* dvec, uint size, uint* dres)
{
    uint x = threadIdx.x;
    if (x<size)
    {
        *dres += dvec[x];
        printf("%u\n", dvec[x]);
    }
    __syncthreads();
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: <filename>\n");
        exit(-1);
    }
    unsigned int log2size, size;
    unsigned int *vec;
    FILE *f = fopen(argv[1], "r");
    fscanf(f, "%d\n", &log2size);
    if (log2size > 10) {
        printf("Size (%u) is too large: size is limited to 2^10\n", log2size);
        exit(-1);
    }
    size = 1 << log2size; // taille du tab
    unsigned int bytes = size * sizeof(unsigned int); // taille des élém du tableau
    vec = (unsigned int *) malloc(bytes); // notre vecteur d'éléments
    assert(vec);
    for (unsigned int i = 0; i < size; i++) {
        fscanf(f, "%u\n", &(vec[i])); // ajout des élem dans vec
    }
    fclose(f);

    uint* dres; // res de la somme on device
    uint res;
    uint* dvec;

    hipError_t err = hipMalloc((void**)&dvec,bytes); //reserve la taille du tab dans la mémoire 
    if (err != hipSuccess){
        printf("%s in %s at line %d\n",
        hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(dvec, vec, bytes, hipMemcpyHostToDevice);
    // dst, src, byte, kind of copy

    err = hipMalloc((void**)&dres, sizeof(int)); // alloc taille res
    if (err != hipSuccess){
        printf("%s in %s at line %d\n",
        hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    kernel<<<1, BSIZE>>>(dvec, size, dres);

    hipMemcpy(&res, dres, sizeof(int), hipMemcpyDeviceToHost);
    // dst, src, byte, kind of copy

    // Objectif : obtenir 34 avec file1
    printf("Somme = %u", res);

    hipFree(dvec);
    hipFree(dres);
}
