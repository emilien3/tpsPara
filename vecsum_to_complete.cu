
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BSIZE 1024


int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: <filename>\n");
        exit(-1);
    }
    unsigned int log2size, size;
    unsigned int *vec;
    FILE *f = fopen(argv[1], "r");
    fscanf(f, "%d\n", &log2size);
    if (log2size > 10) {
        printf("Size (%u) is too large: size is limited to 2^10\n", log2size);
        exit(-1);
    }
    size = 1 << log2size;
    unsigned int bytes = size * sizeof(unsigned int);
    vec = (unsigned int *) malloc(bytes);
    assert(vec);
    for (unsigned int i = 0; i < size; i++) {
        fscanf(f, "%u\n", &(vec[i]));
    }
    fclose(f);
}
